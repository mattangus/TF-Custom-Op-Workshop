#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "tensorflow/core/util/cuda_kernel_helper.h"

template <typename dtype> __global__ void AddKernel(const dtype* a, const dtype* b, dtype* c, int N){
	CUDA_1D_KERNEL_LOOP(index, N)
	{
		c[index] = a[index] + b[index];
	}
}

template <typename dtype>
void launchAddKernel(const dtype* a, const dtype* b, dtype* c, int N) {
	const int kThreadsPerBlock = 1024;
	
	AddKernel<dtype><<<(N + kThreadsPerBlock - 1) / kThreadsPerBlock,
					kThreadsPerBlock>>>(
			a, b, c, N);

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));
}


//forward declaration for all the types needed
#define ADD_KERNEL_TYPE(type)							\
	template void launchAddKernel<type>(				\
		const type* a, const type* b, type* c, int N)	\

ADD_KERNEL_TYPE(int);
ADD_KERNEL_TYPE(float);
ADD_KERNEL_TYPE(double);

#undef ADD_KERNEL_TYPE