#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "tensorflow/core/util/cuda_kernel_helper.h"

template <typename dtype> __global__ void AddKernel(const dtype* a, const dtype* b, dtype* c, int N){

}

template <typename dtype>
void launchAddKernel(const dtype* a, const dtype* b, dtype* c, int N) {
	const int kThreadsPerBlock = 1024;

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));
}
